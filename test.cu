#include "hip/hip_runtime.h"
#include <stdio.h>
#include "Test.cuh"

__global__ void floyd_kernel(int** path_vertices, int k){
	int i =  blockIdx.x;
	int j = threadIdx.x;
	
	int edge = path_vertices[i][j];
	int edge1 = path_vertices[i][k];
	int edge2 = path_vertices[k][j];
	if((edge == -1 && edge1 == -1) || (edge == -1 && edge2 == -1) ){
		path_vertices[i][j] = -1;
	}
	//There is no any path until now 
	else if(edge == -1){
		path_vertices[i][j] = edge2 + edge1;
	}
	//if one of the edge is not exist return current shortest path
	else if(edge1 == -1 || edge2 == -1){
		path_vertices[i][j] =  edge;
	}
	else{
		path_vertices[i][j] = min(edge, edge1 + edge2);
	}
	
}
namspace Wrapper{
	void wrapperfunction(int** shortestPathAllVertices, int numVertives){
 	
	hipMallocManaged(&shortestPathAllVertices, sizeof(int)*numVertices*numVertices);

	for(int k=0; k<numberOfVertices; k++){
		floyd_kernel<<numVertices, numVertices>>(&shortestPathAllVertices, k);
	}	
		
hipDeviceSynchronize();
 }
}